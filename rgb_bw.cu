#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "bitmap_image.hpp"

using namespace std;

__global__ void color_to_grey(int3 *input_image, int3 *output_image, int width, int height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if(col < width && row < height)
    {
        int pos = row * width + col;
        output_image[pos] = { int(input_image[pos].x * 0.30), int(input_image[pos].y * 0.5), int(input_image[pos].z * 0.11)};
    }
}


int main()
{
    bitmap_image bmp("lenna.bmp");

    if(!bmp)
    {
        cerr << "Image not found" << endl;
        exit(1);
    }

    int height = bmp.height();
    int width = bmp.width();
    
    cout << "height " << height << " width " << width << endl;

    //Transform image into vector of doubles
    vector<int3> input_image;
    rgb_t color;
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            bmp.get_pixel(x, y, color);
            input_image.push_back( {color.red, color.green, color.blue} );
        }
    }

    vector<int3> output_image(input_image.size());

    int3 *d_in, *d_out;
    int img_size = (input_image.size() * sizeof(int) * 3);
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);

    hipMemcpy(d_in, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input_image.data(), img_size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(width / 16), ceil(height / 16), 1);
    dim3 dimBlock(16, 16, 1);

    color_to_grey<<< dimGrid , dimBlock >>> (d_in, d_out, width, height);

    hipMemcpy(output_image.data(), d_out, img_size, hipMemcpyDeviceToHost);


    //Set updated pixels
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int pos = x * width + y;
            bmp.set_pixel(x, y, output_image[pos].x, output_image[pos].y, output_image[pos].z);
        }
    }

    bmp.save_image("./grey_scaled.bmp");

    hipFree(d_in);
    hipFree(d_out);
}